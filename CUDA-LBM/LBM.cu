/*
 * 2D flow around a cylinder
 *
 * compile with nvcc -O2 LBM.cu -o LBMcuda
 *
*/

#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>

#define NBLOCKS_X 4
#define NBLOCKS_Y 4
#define NTHREADS_X 32
#define NTHREADS_Y 16


#define PI2 6.28318530718
//====== Flow parameters definition ==============================================
#define MAXITER 500                  // Total number of time iterations.
#define OUTSTEP 10
#define Re 220.0                 // Reynolds number.
#define NX 520                       // Lattice dimensions and populations.
#define NY 180 
#define LY (NY-1.0)
#define Q 9  
#define Q0 3
#define CX (NX/4)                      // Coordinates and radius of the cylinder.
#define CY (NY/2)
#define R (NY/9) 
#define ULB 0.04                  // Velocity in lattice units.


typedef float real_t;
typedef unsigned int uint;

//----- Lattice Constants -------------------------------

__device__ __constant__ real_t C[Q][2] = {
      { 0., 0.},
      { 0.,-1.},
      { 0., 1.},
      {-1., 0.},
      {-1.,-1.},
      {-1., 1.},
      { 1., 0.},
      { 1.,-1.},
      { 1., 1.}
};
__device__ __constant__ int iC[Q][2] = {
      { 0, 0},
      { 0,-1},
      { 0, 1},
      {-1, 0},
      {-1,-1},
      {-1, 1},
      { 1, 0},
      { 1,-1},
      { 1, 1}
};

real_t C_h[Q][2]; //C on host

//noslip = [c.tolist().index((-c[i]).tolist()) for i in range(q)] 
__device__ __constant__ int NOSLIP[Q]={0, 2, 1, 6, 8, 7, 3, 5, 4}; //noslip indexes for C

//i1 = arange(Q)[asarray([ci[0]<0  for ci in c])] # Unknown on right wall.
__device__ __constant__ int I1[Q0] = {3, 4, 5}; // Pops unknown on right wall.
//i2 = arange(Q)[asarray([ci[0]==0 for ci in c])] # Vertical middle.
__device__ __constant__ int I2[Q0] = {0,1,2}; // Vertical middle.
//i3 = arange(Q)[asarray([ci[0]>0  for ci in c])] # Unknown on left wall.
__device__ __constant__ int I3[Q0] = {6, 7, 8}; //Pops Unknown on left wall.


const real_t NULB_h = ULB*(real_t)(R)/(real_t)(Re);
const real_t OMEGA_h = 1.0 / (3.*NULB_h+0.5); // Relaxation parameter.

__device__ __constant__ real_t NULB;
__device__ __constant__ real_t OMEGA; // Relaxation parameter.

//----------------------------------------------------------------------

//========= Functions declaration =================


__host__ __device__ real_t norm2(const real_t * v);

__device__ bool isObstacle(int x, int y);

//convert 2D to 1D array address
// A[i][j]
__device__ __forceinline__ uint i2D(uint i, uint j) {return (NY*i+j);}
//convert 3D to 1D array address
// A[m][i][j]
__device__ __forceinline__ uint i3D(uint m, uint i, uint j) {return (m*NX*NY+NY*i+j);}

__global__ void initialize(real_t * vel, real_t * rho);

__global__ void getEquilibrium(const real_t* rho,
                 const real_t* u,
                 const real_t* t,
                 real_t* feq
                );

__global__ void getHvars(real_t* rho,
            real_t* u,
            const real_t * f
     );

// Right wall: apply outflow condition.
__global__ void outflow(real_t *fin);

// Left wall: compute density from known populations.
__global__ void leftwall(real_t* rho, real_t *u, const real_t* vel, const real_t* fin);

// Left wall: Zou/He boundary condition.
__global__ void zouhe(const real_t* feq, real_t* fin);

// Collision step.
__global__ void collision(const real_t* fin, real_t* fout, const real_t* feq);

//Wall rebound
__global__ void rebound(const real_t * fin, real_t* fout);

// Streaming step.
__global__ void streaming(real_t* fin, const real_t* fout);

//--------------------------------------------------


int main()
{

  // copy NULB_h to NULB on device  
  hipMemcpyToSymbol(HIP_SYMBOL(NULB), &NULB_h, sizeof(real_t), 0, hipMemcpyHostToDevice);  
  // copy OMEGA_h to OMEGA on device
  hipMemcpyToSymbol(HIP_SYMBOL(OMEGA), &OMEGA_h, sizeof(real_t), 0, hipMemcpyHostToDevice);  
  // copy C to C_h on host
  hipMemcpyFromSymbol(&C_h, HIP_SYMBOL(C), 2*Q*sizeof(real_t), 0, hipMemcpyDeviceToHost);

  real_t t[Q];
  real_t* t_d;
  //allocation on device
  hipMalloc((void **)&t_d, Q * sizeof(real_t));

  real_t rho[NX][NY];
  real_t* rho_d;
  //allocation on device
  hipMalloc((void **)&rho_d, NX*NY * sizeof(real_t));

  real_t* vel_d;
  //allocation on device
  hipMalloc((void **)&vel_d, 2*NX*NY * sizeof(real_t));

  real_t u[2][NX][NY];
  real_t* u_d;
  //allocation on device
  hipMalloc((void **)&u_d, 2*NX*NY * sizeof(real_t));

  real_t* feq_d;
  //allocation on device
  hipMalloc((void **)&feq_d, Q*NX*NY * sizeof(real_t));

  real_t* fin_d;
  //allocation on device
  hipMalloc((void **)&fin_d, Q*NX*NY * sizeof(real_t));

  real_t* fout_d;
  //allocation on device
  hipMalloc((void **)&fout_d, Q*NX*NY * sizeof(real_t));
  
   
  t[0]=4./9.;
  for (int iq=1; iq<Q; iq++)
  {
    if (norm2(&C_h[iq][0])<2.)
    {
      t[iq]=1./9.;
    }
    else
    {
      t[iq]=1./36.;
    }
  }
  hipMemcpy((real_t *)t_d,t,Q*sizeof(real_t),hipMemcpyHostToDevice);

  //initial velocity and density setup  
  { 
    dim3 grid(NBLOCKS_X,NBLOCKS_Y,1);
    dim3 threads(NTHREADS_X,NTHREADS_Y,1);
    initialize<<<grid,threads>>>(vel_d,rho_d);
  }
  //-------------------------------

  //equilibrium DF setup
  {
    dim3 grid(NBLOCKS_X,NBLOCKS_Y,1);
    dim3 threads(NTHREADS_X,NTHREADS_Y,1);
    getEquilibrium<<<grid,threads>>>(rho_d,vel_d,t_d,feq_d);
  }
  //-------------------------------

  //copy feq to fin
  hipMemcpy( fin_d, feq_d, sizeof(real_t)*Q*NX*NY,hipMemcpyDeviceToDevice );
  
  //################################################################################
  //###### Main time loop ##########################################################
  
  for (int time=0; time < MAXITER; time++)
  {
    // Right wall: apply outflow condition.
    {
      dim3 grid(1,NBLOCKS_Y,1);
      dim3 threads(1,NTHREADS_Y,1);
      outflow<<<grid,threads>>>(fin_d);
    }
    //---------------------------------

    // Calculate macroscopic density and velocity.
    {
      dim3 grid(NBLOCKS_X,NBLOCKS_Y,1);
      dim3 threads(NTHREADS_X,NTHREADS_Y,1);
      getHvars<<<grid,threads>>>(rho_d,u_d,fin_d);
    }
    //---------------------------------

    
    // Left wall: compute density from known populations.
    {
      dim3 grid(1,NBLOCKS_Y,1);
      dim3 threads(1,NTHREADS_Y,1);
      leftwall<<<grid,threads>>>(rho_d, u_d, vel_d, fin_d);
    }
    //---------------------------------
        
    {
      dim3 grid(NBLOCKS_X,NBLOCKS_Y,1);
      dim3 threads(NTHREADS_X,NTHREADS_Y,1);
      getEquilibrium<<<grid,threads>>>(rho_d,u_d,t_d,feq_d);
    }
    //---------------------------------
    
    // Left wall: Zou/He boundary condition.
    {
      dim3 grid(1,NBLOCKS_Y,1);
      dim3 threads(1,NTHREADS_Y,1);
      zouhe<<<grid,threads>>>(feq_d,fin_d);
    }
    //---------------------------------
    
    // Collision step.
    {
      dim3 grid(NBLOCKS_X,NBLOCKS_Y,1);
      dim3 threads(NTHREADS_X,NTHREADS_Y,1);
      collision<<<grid,threads>>>(fin_d,fout_d,feq_d);
    }
    //---------------------------------

    // Wall "rebound" step.
    {
      dim3 grid(NBLOCKS_X,NBLOCKS_Y,1);
      dim3 threads(NTHREADS_X,NTHREADS_Y,1);
      rebound<<<grid,threads>>>(fin_d,fout_d);
    }
    //---------------------------------
    
    // Streaming step.
    {
      dim3 grid(NBLOCKS_X,NBLOCKS_Y,1);
      dim3 threads(NTHREADS_X,NTHREADS_Y,1);
      streaming<<<grid,threads>>>(fin_d, fout_d);
    }    
    //---------------------------------
    
    // Output.
    if (time % OUTSTEP==0)
    {

      //copy back density and macroscopic velocity from device 
      hipMemcpy(rho,rho_d,NX*NY*sizeof(real_t),hipMemcpyDeviceToHost);
      hipMemcpy(u,u_d,2*NX*NY*sizeof(real_t),hipMemcpyDeviceToHost);

      std::ofstream fout;
      std::stringstream filename("");
      filename <<"out"<<std::setw(5)<<std::setfill('0')<<time<<".csv";
      fout.open(filename.str().c_str(),std::ofstream::out);
      fout << "x,y,rho,ux,uy,|u|" << std::endl;
      for (int x=0; x<NX;x++)
      {
        for (int y=0; y<NY;y++)
        {
          fout << x << "," << y << "," <<rho[x][y] <<"," <<u[0][x][y] << "," << u[1][x][y] << "," 
                << sqrt(norm2(&u[0][x][y])) << std::endl;
        }
      }
      fout.close();
      std::cout << "Step "<<time<<" done.\n";
    }
    
  }
  //=========  free device memory ==========
  hipFree(t_d);

  hipFree(rho_d);

  hipFree(vel_d);

  hipFree(u_d);

  hipFree(feq_d);

  hipFree(fin_d);

  hipFree(fout_d);
  //----------------------------------------
  
}      
//============================================================
//====== functions definition ================================

__host__ __device__ real_t norm2(const real_t * v)
{
  return (v[0]*v[0]+v[1]*v[1]);
} 

__device__ bool isObstacle(int x, int y)
{
  real_t xx[2];
  xx[0]=x-CX; xx[1]=y-CY;
  return norm2(xx)<R*R;
}

// recurrent declarations of threads indexes inside kernels
#define CUDAHEADER_X \
  const uint num_threads_x = gridDim.x*blockDim.x; \
  const uint tstart_x = blockDim.x*blockIdx.x+threadIdx.x;
  
#define CUDAHEADER_Y \
  const uint num_threads_y = gridDim.y*blockDim.y; \
  const uint tstart_y = blockDim.y*blockIdx.y+threadIdx.y;
  
#define CUDAHEADER \
  CUDAHEADER_X \
  CUDAHEADER_Y
  
//-------------------------------------------------------

__global__ void initialize(real_t * vel, real_t * rho)
{
  CUDAHEADER
  
  for (uint x= tstart_x; x<NX; x += num_threads_x)
  {
    for (uint y= tstart_y; y<NY; y += num_threads_y)
    {
      vel[i3D(0,x,y)]=0.;
      if (x>9 && x<20) vel[i3D(0,x,y)]=ULB;
      vel[i3D(1,x,y)]=0.;
      rho[i2D(x,y)]=1.;
    }
  }
}

// Equilibrium distribution function.
__global__ void getEquilibrium(const real_t* rho,
                 const real_t* u,
                 const real_t* t,
                 real_t* feq
                )
{
  CUDAHEADER
  
  real_t cu;
  real_t uxy[2];
  for (uint x= tstart_x; x<NX; x += num_threads_x)
  {
    for (uint y= tstart_y; y<NY; y += num_threads_y)
    {
      uxy[0]=u[i3D(0,x,y)]; uxy[1]=u[i3D(1,x,y)];
      for (uint iq=0; iq<Q;iq++)
      {
        cu = 3.0*(C[iq][0]*uxy[0]+C[iq][1]*uxy[1]);
        feq[i3D(iq,x,y)] = rho[i2D(x,y)]*t[iq]*(1.+cu+0.5*cu*cu-1.5*norm2(uxy));
      }
    }
  }
}

__global__ void getHvars(real_t* rho,
            real_t* u,
            const real_t * f
     )
{
  
  CUDAHEADER
  
  real_t ff;
  for (uint x= tstart_x; x<NX; x += num_threads_x)
  {
    for (uint y= tstart_y; y<NY; y += num_threads_y)
    {
      real_t& rhxy=rho[i2D(x,y)];
      rhxy=0.;
      u[i3D(0,x,y)]=u[i3D(1,x,y)]=0.;
      
      for (uint iq=0; iq<Q;iq++)
      {
        ff=f[i3D(iq,x,y)];
        rhxy += ff;
        u[i3D(0,x,y)] += C[iq][0]*ff;
        u[i3D(1,x,y)] += C[iq][1]*ff;
      }
      u[i3D(0,x,y)] /= rhxy; u[i3D(1,x,y)] /= rhxy;
      
    }
  }
}

// Right wall: apply outflow condition.
__global__ void outflow(real_t *fin)
{

  CUDAHEADER_Y
  
  for (uint y= tstart_y; y<NY; y += num_threads_y)
  {
    for (uint iq=0; iq<Q0;iq++)
    {
      fin[i3D(I1[iq],NX-1,y)] = fin[i3D(I1[iq],NX-2,y)];
    }
  }
}

// Left wall: compute density from known populations.
__global__ void leftwall(real_t* rho, real_t *u, const real_t* vel, const real_t* fin)
{

  CUDAHEADER_Y

  for (uint y= tstart_y; y<NY; y += num_threads_y)
  {
    u[i3D(0,0,y)] =vel[i3D(0,0,y)]; u[i3D(1,0,y)] =vel[i3D(1,0,y)]; 
    real_t &rh0y = rho[i2D(0,y)];
    rh0y = 0.;
    for (uint iq=0; iq<Q0;iq++)
    {
      rh0y += fin[i3D(I2[iq],0,y)] + 2.*fin[i3D(I1[iq],0,y)];
    }
    rh0y /= (1.-u[i3D(0,0,y)]);
  }
}

// Left wall: Zou/He boundary condition.
__global__ void zouhe(const real_t* feq, real_t* fin)
{
  CUDAHEADER_Y
  
  for (uint y= tstart_y; y<NY; y += num_threads_y)
  {
    for (uint iq=0; iq<Q0;iq++)
    {
      fin[i3D(I3[iq],0,y)] = fin[i3D(I1[iq],0,y)] + feq[i3D(I3[iq],0,y)] - feq[i3D(I1[iq],0,y)];
    }
  }
}

// Collision step.
__global__ void collision(const real_t* fin, real_t* fout, const real_t* feq)
{
  CUDAHEADER
  
  for (uint x= tstart_x; x<NX; x += num_threads_x)
  {
    for (uint y= tstart_y; y<NY; y += num_threads_y)
    {
      for (uint iq=0; iq<Q;iq++)
      {
          uint i=i3D(iq,x,y);
          fout[i] = fin[i] - OMEGA * (fin[i] - feq[i]);
      }
    }
  }
}

//Wall rebound
__global__ void rebound(const real_t * fin, real_t* fout)
{
  CUDAHEADER
  
  for (uint x= tstart_x; x<NX; x += num_threads_x)
  {
    for (uint y= tstart_y; y<NY; y += num_threads_y)
    {
      if (isObstacle(x,y))
      {
        for (uint iq=0; iq<Q;iq++)
        {
          fout[i3D(iq,x,y)] = fin[i3D(NOSLIP[iq],x,y)];
        }
      }
    }
  }
}

// Streaming step.
__global__ void streaming(real_t* fin, const real_t* fout)
{
  CUDAHEADER
  int xout,yout;
  
  for (int x= tstart_x; x<NX; x += num_threads_x)
  {
    for (int y= tstart_y; y<NY; y += num_threads_y)
    {
      fin[i3D(0,x,y)]=fout[i3D(0,x,y)];
      for (int iq=1; iq<Q;iq++)
      {
          //handle periodic conditions
          xout = ((x + iC[iq][0])+NX) % NX; 
          yout = ((y + iC[iq][1])+NY) % NY;
          fin[i3D(iq,xout,yout)]=fout[i3D(iq,x,y)];
      }
    }
  }
}  


